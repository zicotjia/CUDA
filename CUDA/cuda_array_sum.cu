#include <iostream>
#include <hip/hip_runtime.h>

__device__ unsigned long long d_result = 0;

__global__ void addition(unsigned const int* arr, int size) {
    u_int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        atomicAdd(&d_result, arr[idx]);
    }
}

__global__ void optimizedAddition(unsigned const int* arr, int size) {
    extern __shared__ unsigned int sharedArr[];

    // Version 1: less optimal
//    u_int arr_idx = threadIdx.x + blockIdx.x * blockDim.x;
//    if (arr_idx < size) {
//        sharedArr[threadIdx.x] = arr[arr_idx];
//    } else {
//        sharedArr[threadIdx.x] = 0;
//    }

    // Version 2: Optimized to use all threads to do 1st round sum
    // Preemptively do 1st round sum here
    u_int arr_idx = threadIdx.x + blockIdx.x * blockDim.x * 2;

    unsigned int sum = 0;
    if (arr_idx < size) {
        sum = arr[arr_idx];
    }
    if (arr_idx + blockDim.x < size) {
        sum += arr[arr_idx + blockDim.x];
    }

    sharedArr[threadIdx.x] = sum;

    __syncthreads();

    for (unsigned int endIndex = blockDim.x / 2; endIndex > 0; endIndex /= 2) {
        if (threadIdx.x < endIndex) {
            sharedArr[threadIdx.x] += sharedArr[threadIdx.x + endIndex];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(&d_result, sharedArr[0]);
    }
}

unsigned long long additionCpu(unsigned const int* arr, int size) {
    unsigned long long sum = 0;
    for (int i = 0; i < size; i++) {
        sum += arr[i];
    }
    return sum;
}

int main() {
    constexpr unsigned int BLOCK_SIZE = 256;
    constexpr unsigned long long N = 10'000'000;

    // Malloc for host array
    auto* h_arr = static_cast<unsigned int *>(malloc(sizeof(unsigned int) * N));

    // Initialise host array Elements
    for (int i = 0; i < N; i++) {
        h_arr[i] = 1;
    }

    // Malloc for device array
    unsigned int* d_arr;
    hipMalloc((void**) &d_arr, N * sizeof(unsigned int));

    // Copy host array to device array
    hipMemcpy(d_arr, h_arr, N * sizeof(unsigned int), hipMemcpyHostToDevice);

    // define block dimension
    const dim3 block_dim = { BLOCK_SIZE, 1, 1 };

    // define grid dimension
    // For non-optimisation and optimisation version 1;
    // const dim3 gridDim = { (N + BLOCK_SIZE * 2 - 1) / (2 * BLOCK_SIZE), 1, 1 };

    // For optimisation version 2
    const dim3 grid_dim = { (N + BLOCK_SIZE * 2 - 1) / (2 * BLOCK_SIZE), 1, 1 };

    // define shared memory size
    const int sharedMemorySize = BLOCK_SIZE * sizeof(unsigned int);

    // set device result initial value
    hipMemset(&d_result, 0, sizeof(unsigned long long));

    // invoke kernel
    optimizedAddition<<<grid_dim, block_dim, sharedMemorySize>>>(d_arr, N);
    hipDeviceSynchronize();

    // define host result
    unsigned long long h_result = 0.0f;

    // copy device result to host result
    hipMemcpyFromSymbol(&h_result, HIP_SYMBOL(d_result), sizeof(unsigned long long));

    // CPU version for comparison
    unsigned long long cpuResult = additionCpu(h_arr, N);

    std::cout << "GPU SUM: " << h_result << std::endl;
    std::cout << "CPU SUM: " << cpuResult << std::endl;

    // Free memory
    hipFree(d_arr);
    free(h_arr);
    return 0;
}
